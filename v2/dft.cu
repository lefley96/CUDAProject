#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
 
#define THREADS_PER_BLOCK 512

__constant__ float EXPCONST = (-2.0 * 3.141592653);
 
__device__ __forceinline__ hipComplex my_cexpf(hipComplex z) {
    hipComplex res;
    float t = expf(z.x);
 
    sincosf(z.y, &res.y, &res.x);
    res.x *= t;
    res.y *= t;
	
    return res;
}
 
__global__ void dftKernel(hipComplex *input, hipComplex *output, int size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < size) {
        hipComplex tempSum = make_hipComplex(0.0, 0.0);
        for (int i = 0; i < size; i++) {
            tempSum = hipCaddf(tempSum, hipCmulf(input[i], my_cexpf(make_hipComplex(0.0, (EXPCONST * i * idx / size)))));
        }
        output[idx] = tempSum;
    }
}
 
extern "C" void dft(float* samples, int size, float* real, float* imag) {
	hipComplex* complex_samples = (hipComplex*) malloc(size * sizeof(hipComplex));

	int i;
	for (i = 0; i < size; i++) {
		complex_samples[i] = make_hipComplex(samples[i], 0);
	}

	hipComplex *d_input, *d_output;
 
	hipMalloc((void **) &d_input, size * sizeof(hipComplex));
	hipMalloc((void **) &d_output, size * sizeof(hipComplex));

	hipMemcpy(d_input, complex_samples, size * sizeof(hipComplex), hipMemcpyHostToDevice);

	dftKernel<<<(size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_input, d_output, size);

	hipDeviceSynchronize();

	hipMemcpy(complex_samples, d_output, size * sizeof(hipComplex), hipMemcpyDeviceToHost);

	for (i = 0; i < size; i++) {
		real[i] = hipCrealf(complex_samples[i]);
		imag[i] = hipCimagf(complex_samples[i]);
	}

	hipFree(d_input);
	hipFree(d_output);
}

/*int main() {
	float a[8] = {92, 79, 68, 32, 16, 40, 7, 87};
	float r[8];
	float i[8];
	
	dft(a, 8, r, i);
 
	printf("output\n");

	int j;
	for (j = 0; j < 8; j++) {
        	printf("%d: %f %f %f\n", j + 1, a[j], r[j], i[j]);
	}

	return 0;
}*/

