#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

__constant__ float EXPCONST = (-2.0 * 3.141592653);

__device__ __forceinline__ hipComplex my_cexpf(hipComplex z) {
    hipComplex res;
    float t = expf(z.x);

    sincosf(z.y, &res.y, &res.x);
    res.x *= t;
    res.y *= t;

    return res;
}
 
__global__ void dftKernel(hipComplex *input, hipComplex *output, int size) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < size) {
        hipComplex tempSum = make_hipComplex(0.0, 0.0);

        for (int i = 0; i < size; i++) {
            tempSum = hipCaddf(tempSum, hipCmulf(input[i], my_cexpf(make_hipComplex(0.0, (EXPCONST * i * idx / size)))));
        }

        output[idx] = tempSum;
    }
}

int chooseAndSetBestDevice() {
    int num_devices, device;
    int max_multiprocessors = 0, max_device = 0;
    
    hipGetDeviceCount(&num_devices);
	
    if (num_devices > 1) {          
		for (device = 0; device < num_devices; device++) {
			hipDeviceProp_t properties;
			hipGetDeviceProperties(&properties, device);
			
			if (max_multiprocessors < properties.multiProcessorCount) {
				max_multiprocessors = properties.multiProcessorCount;
				max_device = device;
			}
		}
    }
	
    hipSetDevice(max_device);
    return max_device; 
}

int getThreadsPerBlock(int currentDevice) {
    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, currentDevice);
    return properties.maxThreadsPerBlock;
}
 
extern "C" void dft(float* samples, int size, float* real, float* imag) {
    int currentDevice = chooseAndSetBestDevice();
    int THREADS_PER_BLOCK = getThreadsPerBlock(currentDevice);

    hipComplex* complex_samples = (hipComplex*) malloc(size * sizeof(hipComplex));

    int i;
    for (i = 0; i < size; i++) {
		complex_samples[i] = make_hipComplex(samples[i], 0);
    }

    hipComplex *d_input, *d_output;
 
    hipMalloc((void **) &d_input, size * sizeof(hipComplex));
    hipMalloc((void **) &d_output, size * sizeof(hipComplex));

    hipMemcpy(d_input, complex_samples, size * sizeof(hipComplex), hipMemcpyHostToDevice);

    dftKernel<<<(size + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_input, d_output, size);

    hipDeviceSynchronize();

    hipMemcpy(complex_samples, d_output, size * sizeof(hipComplex), hipMemcpyDeviceToHost);

    for (i = 0; i < size; i++) {
        real[i] = hipCrealf(complex_samples[i]);
        imag[i] = hipCimagf(complex_samples[i]);
    }

    hipFree(d_output);
    hipFree(d_input);
    free(complex_samples);
}

